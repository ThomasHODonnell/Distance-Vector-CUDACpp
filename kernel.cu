#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "header.cuh"

#include <stdio.h> 
#include <math.h>

#define N 2048
#define TPB 1024 // change to 1025+ to throw sync error (no more than 1024 threads on my system)

const int GRIDSIZE = (N + TPB - 1) / TPB;

float scale(int in, int size) { return ((float)in) / (size - 1); }

__device__ float distance(float x1, float x2) { return sqrt(pow(x2 - x1, 2)); }

__global__ void distanceKernel(float* d_out, float* d_in, float ref) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]); 
}

int main() {

	deviceProps();

	const float ref = 0.5f; 

	float* in = 0; 
	float* out = 0; 

	hipError_t inMalErr = hipMallocManaged(&in, N * sizeof(float));
	if (inMalErr != hipSuccess) { printf("Input Array Malloc Error: code %d - %s.\n", hipError_t(inMalErr), hipGetErrorString(inMalErr)); return -1; }
	hipError_t outMalErr = hipMallocManaged(&out, N * sizeof(float));
	if (outMalErr != hipSuccess) {
		printf("Output Array Malloc Error: code %d - %s.\n", hipError_t(outMalErr), hipGetErrorString(outMalErr)); return -1; }

	for (int i = 0; i < N; ++i) in[i] = scale(i, N);

	distanceKernel << <GRIDSIZE, TPB >> > (out, in, ref);
	hipError_t syncErr = hipGetLastError();
	hipError_t asyncErr = hipDeviceSynchronize();
	if (syncErr != hipSuccess) { printf("Sync Kernel Error: code %d - %s.\n", hipError_t(syncErr), hipGetErrorString(syncErr)); return -1; }
	if (asyncErr != hipSuccess) {
		printf("Async Kernel Error: code %d - %s.\n", hipError_t(asyncErr), hipGetErrorString(asyncErr)); return -1; }

	hipFree(in);
	hipFree(out);

	return 0;
}